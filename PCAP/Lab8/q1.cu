#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 3 // Matrix size (NxN)

// Kernel for row-wise computation
__global__ void addRows(int *A, int *B, int *C, int n) {
    int row = blockIdx.x;
    if (row < n) {
        for (int j = 0; j < n; j++) {
            C[row * n + j] = A[row * n + j] + B[row * n + j];
        }
    }
}

// Kernel for column-wise computation
__global__ void addCols(int *A, int *B, int *C, int n) {
    int col = blockIdx.x;
    if (col < n) {
        for (int i = 0; i < n; i++) {
            C[i * n + col] = A[i * n + col] + B[i * n + col];
        }
    }
}

// Kernel for element-wise computation
__global__ void addElements(int *A, int *B, int *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        int idx = row * n + col;
        C[idx] = A[idx] + B[idx];
    }
}

// Function to print the matrix
void printMatrix(int *M, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", M[i * n + j]);
        }
        printf("\n");
    }
}

int main() {
    int size = N * N * sizeof(int);
    int A[N * N], B[N * N], C[N * N];

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        A[i] = i;
        B[i] = i * 2;
    }

    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    printf("Original Matrices:\n");
    printMatrix(A, N);
    printf("\n+\n");
    printMatrix(B, N);
    printf("\n=\n");

    // Case (a): Each row computed by one thread
    addRows<<<N, 1>>>(d_A, d_B, d_C, N);

    // Case (b): Each column computed by one thread
    // addCols<<<N, 1>>>(d_A, d_B, d_C, N);

    // Case (c): Each element computed by one thread
    // dim3 threadsPerBlock2D(16, 16);
    // dim3 numBlocks2D((N + 15) / 16, (N + 15) / 16);
    // addElements<<<numBlocks2D, threadsPerBlock2D>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    printMatrix(C, N);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
